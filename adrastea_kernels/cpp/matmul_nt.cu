
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// output = lhs * rhs^T; lhs = (m, p); rhs = (n, p); output = (m, n)
// this only exists as a trivial reference point for optimized kernels.
extern "C" __global__ void
matmul_nt(__half* output, __half* lhs, __half* rhs, int m, int p, int n, float beta = 0.0f) {
  int c = blockIdx.x * blockDim.x + threadIdx.x;
  int r = blockIdx.y * blockDim.y + threadIdx.y;
  if (r < m && c < n) {
    float sum = 0;
    for (int i = 0; i < p; i++) {
      sum += __half2float(lhs[r * p + i]) * __half2float(rhs[c * p + i]);
    }
    output[r * n + c] = sum + beta * __half2float(output[r * n + c]);
  }
}
