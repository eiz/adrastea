
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// row-wise rms normalization
// 1 block per row, x = row, 8 warps per block
extern "C" __global__ void rms_norm(__half* output,
                                    __half* input,
                                    __half* weights,
                                    int h,
                                    int w,
                                    float eps) {
  int row = blockIdx.x;
  int tid = threadIdx.x;
  int row_idx = row * w;
  int warp_id = tid / 32;
  bool warp_leader = (tid % 32) == 0;
  __shared__ float s_rms_inv;
  __shared__ float s_warp_reduced[8];
  float sum_val = 0.0f;
  // sum_sq: thread reduction
  for (int i = tid; i < w; i += blockDim.x) {
    float val = __half2float(input[row_idx + i]);
    sum_val += val * val;
  }
  __syncthreads();
  // sum_sq: warp reduction
  for (int offset = 16; offset > 0; offset /= 2) {
    float other_val = __shfl_xor_sync(~0, sum_val, offset);
    sum_val += other_val;
  }
  if (warp_leader) {
    s_warp_reduced[warp_id] = sum_val;
  }
  // sum_sq: block reduction
  __syncthreads();
  if (warp_id == 0) {
    sum_val = (tid < 8) ? s_warp_reduced[tid] : 0.0f;
    for (int offset = 4; offset > 0; offset /= 2) {
      float other_val = __shfl_xor_sync(~0, sum_val, offset);
      sum_val += other_val;
    }
    if (warp_leader) {
      s_rms_inv = rsqrt((sum_val / w) + eps);
    }
  }
  __syncthreads();
  float rms_inv = s_rms_inv;
  for (int i = tid; i < w; i += blockDim.x) {
    float val = __half2float(input[row_idx + i]);
    output[row_idx + i] = weights[i] * __float2half(val * rms_inv);
  }
}
