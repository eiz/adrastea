#include "hip/hip_runtime.h"
#include "compat.h"

// the simplest quantized right side kernel. reference.
extern "C" __global__ void matmul_nt_fp16u8(half* __restrict__ output,
                                            half const* __restrict__ lhs,
                                            uint8_t const* __restrict__ rhs,
                                            half const* __restrict__ scales,
                                            int m,
                                            int p,
                                            int n,
                                            int block_size,
                                            float beta = 0.0f) {
  int c = BLOCK_IDX_X * BLOCK_DIM_X + THREAD_IDX_X;
  int r = BLOCK_IDX_Y * BLOCK_DIM_Y + THREAD_IDX_Y;
  int block_count = p / block_size;
  if (r < m && c < n) {
    float sum = 0;
    for (int i = 0; i < p; i++) {
      sum += __half2float(lhs[r * p + i]) * (float(rhs[c * p + i]) - 127.5) *
             __half2float(scales[c * block_count + i / block_size]);
    }
    output[r * n + c] = sum + beta * __half2float(output[r * n + c]);
  }
}
