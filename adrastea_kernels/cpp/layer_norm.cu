#include "hip/hip_runtime.h"
#include "compat.h"

// row-wise layer normalization
// 1 block per row, x = row, 8 warps per block
// TODO strides kill the performance, refuxor later
extern "C" __global__ void layer_norm(__half* output,
                                      __half* input,
                                      __half* weights,
                                      __half* bias,
                                      int length_x,
                                      int length_y,
                                      int stride_ox,
                                      int stride_oy,
                                      int stride_ix,
                                      int stride_iy,
                                      float eps) {
  int row = BLOCK_IDX_X;
  int tid = THREAD_IDX_X;
  int in_row_idx = row * stride_iy;
  int out_row_idx = row * stride_oy;
  int warp_id = tid / 32;
  bool warp_leader = (tid % 32) == 0;
  __shared__ float s_mean;
  __shared__ float s_stddev;
  __shared__ float s_warp_reduced[8];
  float sum_val = 0.0f;
  // sum: thread reduction
  for (int i = tid; i < length_x; i += BLOCK_DIM_X) {
    float val = __half2float(input[in_row_idx + i * stride_ix]);
    sum_val += val;
  }
  __syncthreads();
  // sum: warp reduction
  for (int offset = 16; offset > 0; offset /= 2) {
    float other_val = __shfl_xor_sync(~0, sum_val, offset);
    sum_val += other_val;
  }
  if (warp_leader) {
    s_warp_reduced[warp_id] = sum_val;
  }
  // sum: block reduction
  __syncthreads();
  if (warp_id == 0) {
    sum_val = (tid < 8) ? s_warp_reduced[tid] : 0.0f;
    for (int offset = 4; offset > 0; offset /= 2) {
      float other_val = __shfl_xor_sync(~0, sum_val, offset);
      sum_val += other_val;
    }
    if (warp_leader) {
      s_mean = sum_val / length_x;
    }
  }
  __syncthreads();
  sum_val = 0.0f;
  // mean diff: thread reduction
  for (int i = tid; i < length_x; i += BLOCK_DIM_X) {
    float val = __half2float(input[in_row_idx + i * stride_ix]);
    sum_val += (val - s_mean) * (val - s_mean);
  }
  __syncthreads();
  // mean diff: warp reduction
  for (int offset = 16; offset > 0; offset /= 2) {
    float other_val = __shfl_xor_sync(~0, sum_val, offset);
    sum_val += other_val;
  }
  if (warp_leader) {
    s_warp_reduced[warp_id] = sum_val;
  }
  // mean diff: block reduction
  __syncthreads();
  if (warp_id == 0) {
    sum_val = (tid < 8) ? s_warp_reduced[tid] : 0.0f;
    for (int offset = 4; offset > 0; offset /= 2) {
      float other_val = __shfl_xor_sync(~0, sum_val, offset);
      sum_val += other_val;
    }
    if (warp_leader) {
      s_stddev = sqrt(sum_val / length_x + eps);
    }
  }
  __syncthreads();
  for (int i = tid; i < length_x; i += BLOCK_DIM_X) {
    output[out_row_idx + i * stride_ox] =
        (__half2float(input[in_row_idx + i * stride_ix]) - s_mean) / s_stddev *
            __half2float(weights[i]) +
        __half2float(bias[i]);
  }
}
