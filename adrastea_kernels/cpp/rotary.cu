#include "hip/hip_runtime.h"
#include "compat.h"

extern "C" __global__ void rotary(__half* output,
                                  __half* input,
                                  int h,
                                  int w,
                                  int n_heads,
                                  int pos_offset = 0,
                                  float theta = 10000.0) {
  int r = BLOCK_IDX_Y * BLOCK_DIM_Y + THREAD_IDX_Y;
  int c = 2 * (BLOCK_IDX_X * BLOCK_DIM_X + THREAD_IDX_X);
  int head_dim = w / n_heads;
  int head_c = c % head_dim;

  if (r < h && c < w) {
    float angle = (pos_offset + r) / powf(theta, float(head_c) / head_dim);
    float real = __half2float(input[r * w + c]);
    float imag = __half2float(input[r * w + c + 1]);
    float a_cos = cosf(angle);
    float a_sin = sinf(angle);
    output[r * w + c] = __float2half(real * a_cos - imag * a_sin);
    output[r * w + c + 1] = __float2half(real * a_sin + imag * a_cos);
  }
}
