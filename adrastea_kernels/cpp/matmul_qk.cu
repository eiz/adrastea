
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_math_constants.h>

// q * k^T
// output = (n_heads, seq_len_new, seq_len)
// lhs = (seq_len_new, dim)
// rhs = (seq_len, dim)
// we have to do the swizzle to regroup by heads here
// basically we have the above as inputs but we want to access them like
// lhs = (n_heads, seq_len_new, head_dim)
// rhs = (n_heads, head_dim, seq_len)
// grid x/y are row/col indices for the output, z is the head index
extern "C" __global__ void matmul_qk(__half* output,
                                     __half* lhs,
                                     __half* rhs,
                                     int seq_len_new,
                                     int seq_len,
                                     int dim,
                                     int n_heads,
                                     int start_pos) {
  // TODO: write a tiled kernel for this. only for testing accuracy.
  // probably write a cuBLAS path too which will need to materialize the
  // transposes.
  int c = blockIdx.x * blockDim.x + threadIdx.x;
  int r = blockIdx.y * blockDim.y + threadIdx.y;
  int head = blockIdx.z;
  int head_dim = dim / n_heads;
  bool masked = c > (r + start_pos);
  if (r < seq_len_new && c < seq_len) {
    if (masked) {
      output[head * seq_len_new * seq_len + r * seq_len_new + c] = -HIP_INF_F;
    } else {
      float sum = 0;
      for (int i = 0; i < head_dim; i++) {
        sum += __half2float(lhs[r * dim + head * head_dim + i]) *
               __half2float(rhs[c * dim + head * head_dim + i]);
      }
      output[head * seq_len_new * seq_len + r * seq_len_new + c] = sum / sqrt(float(head_dim));
    }
  }
}
